#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstring>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>

#include "SimulationNBodyCUDA.hpp"
#include "core/Bodies.hpp"

// static dim3 blocksPerGrid = {1};
// static dim3 threadsPerBlock = {1024};
// static float *d_qx, *d_qy, *d_qz, *d_m;
// static accAoS_t<float> *d_accelerations;

//static dim3 blocksPerGrid = {60} ;
//static dim3 threadsPerBlock = {1024};

__constant__ float softSquared;
__constant__ float gravity;


SimulationNBodyCUDA::SimulationNBodyCUDA(const unsigned long nBodies, const std::string &scheme, const float soft,
                                         const unsigned long randInit)
    : SimulationNBodyInterface(nBodies, scheme, soft, randInit)
{

    const float hsoft = this->soft * this->soft;
    
    hipMemcpyToSymbol(HIP_SYMBOL(softSquared), &hsoft, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(gravity), &this->G, sizeof(float), 0, hipMemcpyHostToDevice);

    const unsigned long arraySize = sizeof(float) * nBodies;
    const unsigned long accSize = sizeof(struct accAoS_t<float>) * nBodies;
    this->flopsPerIte = 19.f * (float)this->getBodies().getN() * (float)this->getBodies().getN() + 3.f * (float)this->getBodies().getN();
    this->accelerations.resize(this->getBodies().getN());
    hipMalloc(&this->d_qx, arraySize);
    hipMalloc(&this->d_qy, arraySize);
    hipMalloc(&this->d_qz, arraySize);
    hipMalloc(&this->d_m, arraySize);
    hipMalloc(&this->d_accelerations, accSize);
}

 SimulationNBodyCUDA::~SimulationNBodyCUDA() {
      hipFree(d_qx);
      hipFree(d_qy);
      hipFree(d_qz);
      hipFree(d_m);
      hipFree(d_accelerations);
    }


void SimulationNBodyCUDA::initIteration()
{
    for (unsigned long iBody = 0; iBody < this->getBodies().getN(); iBody++) {
        this->accelerations[iBody].ax = 0.f;
        this->accelerations[iBody].ay = 0.f;
        this->accelerations[iBody].az = 0.f;
    }
}

static __global__ void computeBodiesAcceleration(const unsigned long nBodies,
                                          float *qx, float *qy, float *qz, float *m, accAoS_t<float> *accelerations)
{
        int x = blockDim.x * blockIdx.x + threadIdx.x;

        if(x > nBodies){
            return;
        }
        
        
        float ax = 0, ay = 0, az = 0;
        for (unsigned long jBody = 0; jBody < nBodies; jBody++) {

            // All forces of bodies of indexes lower than the current one have already been added to current body's
            // accel skiping.
            const float rijx = qx[jBody]- qx[x]; // 1 flop
            const float rijy = qy[jBody]- qy[x]; // 1 flop
            const float rijz = qz[jBody] - qz[x]; // 1 flop

            // compute the || rij ||² distance between body i and body j
            float rijSquared = rijx * rijx + rijy * rijy + rijz * rijz; // 5 flops
            // compute e²
            rijSquared += softSquared; // 1 flop

            const float pow = rsqrtf(rijSquared); // 1 flops

            // compute the acceleration value between body i and body j: || ai || = G.mj / (|| rij ||² + e²)^{3/2}
            const float ai = m[jBody] * (pow * pow * pow); // 3 flops

            // add the acceleration value into the acceleration vector: ai += || ai ||.rij
            ax += ai * rijx; // 2 flops
            ay += ai * rijy; // 2 flops
            az += ai * rijz; // 2 flops
            
            // accelerations[x].ax += ai *rijx;            
            // accelerations[x].ay += ai * rijy;            
            // accelerations[x].az += ai * rijz;            

            // Adding acceleration forces to the j body as well.
        }

        accelerations[x].ax = ax * gravity;
        accelerations[x].ay = ay * gravity;
        accelerations[x].az = az * gravity;
}

void SimulationNBodyCUDA::computeOneIteration()
{
    this->initIteration();
    //const float softSquared = this->soft * this->soft;
    const unsigned long nBodies = this->getBodies().getN();
    dataSoA_t<float> h_bodies = this->getBodies().getDataSoA();
    const unsigned long arraySize = sizeof(float) * nBodies;
    const unsigned long accSize = sizeof(struct accAoS_t<float>) * nBodies;

    // hipMalloc(&this->d_qx, arraySize);
    // hipMalloc(&this->d_qy, arraySize);
    // hipMalloc(&this->d_qz, arraySize);
    // hipMalloc(&this->d_m, arraySize);
    // hipMalloc(&this->d_accelerations, accSize);
    hipMemset(d_accelerations, 0, accSize);
    hipMemset(d_qx, 0, arraySize);
    hipMemset(d_qy, 0, arraySize);
    hipMemset(d_qz, 0, arraySize);
    hipMemset(d_m, 0, arraySize);
    

    hipMemcpy(d_qx, &(h_bodies.qx[0]), arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_qy, &(h_bodies.qy[0]), arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_qz, &(h_bodies.qz[0]), arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_m, &(h_bodies.m[0]), arraySize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock = {512};
    dim3 blocksPerGrid = {(nBodies+threadsPerBlock.x -1)/threadsPerBlock.x};
    

    computeBodiesAcceleration<<< blocksPerGrid,threadsPerBlock >>>(nBodies, d_qx, d_qy, d_qz, d_m, d_accelerations);

    hipDeviceSynchronize();

    hipMemcpy(this->accelerations.data(), d_accelerations, accSize, hipMemcpyDeviceToHost);

    // hipFree(d_qx);
    // hipFree(d_qy);
    // hipFree(d_qz);
    // hipFree(d_m);
    // hipFree(d_accelerations); 
    
    // time integration
    this->bodies.updatePositionsAndVelocities(this->accelerations, this->dt);
}
