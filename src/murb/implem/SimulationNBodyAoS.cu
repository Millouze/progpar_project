#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstring>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

#include "SimulationNBodyAoS.hpp"
#include "core/Bodies.hpp"

static dim3 blocksPerGrid = {60};
static dim3 threadsPerBlock = {1024};
dataAoS_t<float> *d_bodies;
accAoS_t<float> *d_accelerations;

SimulationNBodyAoS::SimulationNBodyAoS(const unsigned long nBodies, const std::string &scheme, const float soft,
                                         const unsigned long randInit)
    : SimulationNBodyInterface(nBodies, scheme, soft, randInit)
{
    this->flopsPerIte = 20.f * (float)this->getBodies().getN() * (float)this->getBodies().getN();
    this->accelerations.resize(this->getBodies().getN());
}

void SimulationNBodyAoS::initIteration()
{
    for (unsigned long iBody = 0; iBody < this->getBodies().getN(); iBody++) {
        this->accelerations[iBody].ax = 0.f;
        this->accelerations[iBody].ay = 0.f;
        this->accelerations[iBody].az = 0.f;
    }
}

__global__ void computeBodiesAcceleration(const unsigned long nBodies, const float softSquared, const float G,
                                          dataAoS_t<float> *d, accAoS_t<float> *accelerations)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;

        if(x > nBodies){
            return;
        }
        
        float ax = accelerations[x].ax, ay = accelerations[x].ay,
              az = accelerations[x].az;
        for (unsigned long jBody = 0; jBody < nBodies; jBody++) {

            // All forces of bodies of indexes lower than the current one have already been added to current body's
            // accel skiping.
            const float rijx = d[jBody].qx - d[x].qx; // 1 flop
            const float rijy = d[jBody].qy - d[x].qy; // 1 flop
            const float rijz = d[jBody].qz - d[x].qz; // 1 flop

            // compute the || rij ||² distance between body i and body j
            float rijSquared = rijx * rijx + rijy * rijy + rijz * rijz; // 5 flops
            // compute e²
            rijSquared += softSquared;

            const float pow = rsqrtf(rijSquared); // 2 flops

            // compute the acceleration value between body i and body j: || ai || = G.mj / (|| rij ||² + e²)^{3/2}
            const float ai = G * d[jBody].m * (pow * pow * pow); // 3 flops

            // add the acceleration value into the acceleration vector: ai += || ai ||.rij
            ax += ai * rijx; // 2 flops
            ay += ai * rijy; // 2 flops
            az += ai * rijz; // 2 flops

            // Adding acceleration forces to the j body as well.
        }

        accelerations[x].ax = ax;
        accelerations[x].ay = ay;
        accelerations[x].az = az;

}

void SimulationNBodyAoS::computeOneIteration()
{
    this->initIteration();
    const float softSquared = this->soft * this->soft;
    const unsigned long nBodies = this->getBodies().getN();
    const std::vector<dataAoS_t<float>> h_bodies = this->getBodies().getDataAoS();
    std::vector<accAoS_t<float>> h_accelerations = this->accelerations;

    hipMalloc(&d_bodies, sizeof(struct dataAoS_t<float>) * nBodies);
    hipMalloc(&d_accelerations, sizeof(struct accAoS_t<float>) * nBodies);

    hipMemcpy(d_bodies, h_bodies.data(), sizeof(struct dataAoS_t<float>) * nBodies, hipMemcpyHostToDevice);

    computeBodiesAcceleration<<< blocksPerGrid,threadsPerBlock >>>(this->getBodies().getN(), softSquared, this->G, d_bodies, d_accelerations);

    
    hipGetLastError(); 
    hipDeviceSynchronize();
    
    hipMemcpy(this->accelerations.data(), d_accelerations, sizeof(struct accAoS_t<float>) * nBodies, hipMemcpyDeviceToHost);

    hipFree(d_bodies);
    hipFree(d_accelerations);
    // time integration
    this->bodies.updatePositionsAndVelocities(this->accelerations, this->dt);
} 
