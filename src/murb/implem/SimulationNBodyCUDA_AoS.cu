#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstring>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>

#include "SimulationNBodyCUDA_AoS.hpp"
#include "core/Bodies.hpp"

//static dim3 blocksPerGrid = {60};
//static dim3 threadsPerBlock = {1024};
static dataAoS_t<float> *d_bodies;
static accAoS_t<float> *d_accelerations;

SimulationNBodyCUDA_AoS::SimulationNBodyCUDA_AoS(const unsigned long nBodies, const std::string &scheme, const float soft,
                                         const unsigned long randInit)
    : SimulationNBodyInterface(nBodies, scheme, soft, randInit)
{
    this->flopsPerIte = 20.f * (float)this->getBodies().getN() * (float)this->getBodies().getN();
    this->accelerations.resize(this->getBodies().getN());
}

void SimulationNBodyCUDA_AoS::initIteration()
{
    for (unsigned long iBody = 0; iBody < this->getBodies().getN(); iBody++) {
        this->accelerations[iBody].ax = 0.f;
        this->accelerations[iBody].ay = 0.f;
        this->accelerations[iBody].az = 0.f;
    }
}

__global__ void computeBodiesAcceleration(const unsigned long nBodies, const float softSquared, const float G,
                                          dataAoS_t<float> *d, accAoS_t<float> *accelerations)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;

        if(x > nBodies){
            return;
        }
        
        float ax = accelerations[x].ax, ay = accelerations[x].ay,
              az = accelerations[x].az;
        for (unsigned long jBody = 0; jBody < nBodies; jBody++) {

            // All forces of bodies of indexes lower than the current one have already been added to current body's
            // accel skiping.
            const float rijx = d[jBody].qx - d[x].qx; // 1 flop
            const float rijy = d[jBody].qy - d[x].qy; // 1 flop
            const float rijz = d[jBody].qz - d[x].qz; // 1 flop

            // compute the || rij ||² distance between body i and body j
            float rijSquared = rijx * rijx + rijy * rijy + rijz * rijz; // 5 flops
            // compute e²
            rijSquared += softSquared;

            const float pow = rsqrtf(rijSquared); // 2 flops

            // compute the acceleration value between body i and body j: || ai || = G.mj / (|| rij ||² + e²)^{3/2}
            const float ai = G * d[jBody].m * (pow * pow * pow); // 3 flops

            // add the acceleration value into the acceleration vector: ai += || ai ||.rij
            ax += ai * rijx; // 2 flops
            ay += ai * rijy; // 2 flops
            az += ai * rijz; // 2 flops

            // Adding acceleration forces to the j body as well.
        }

        accelerations[x].ax = ax;
        accelerations[x].ay = ay;
        accelerations[x].az = az;

}

void SimulationNBodyCUDA_AoS::computeOneIteration()
{
    this->initIteration();
    const float softSquared = this->soft * this->soft;
    const unsigned long nBodies = this->getBodies().getN();
    const std::vector<dataAoS_t<float>> h_bodies = this->getBodies().getDataAoS();
    std::vector<accAoS_t<float>> h_accelerations = this->accelerations;

    hipMalloc(&d_bodies, sizeof(struct dataAoS_t<float>) * nBodies);
    hipMalloc(&d_accelerations, sizeof(struct accAoS_t<float>) * nBodies);

    hipMemcpy(d_bodies, h_bodies.data(), sizeof(struct dataAoS_t<float>) * nBodies, hipMemcpyHostToDevice);

    dim3 blocksPerGrid = {(nBodies+1023)/1024};
    dim3 threadsPerBlock = {1024};

    computeBodiesAcceleration<<< blocksPerGrid,threadsPerBlock >>>(this->getBodies().getN(), softSquared, this->G, d_bodies, d_accelerations);

    
    hipGetLastError(); 
    hipDeviceSynchronize();
    
    hipMemcpy(this->accelerations.data(), d_accelerations, sizeof(struct accAoS_t<float>) * nBodies, hipMemcpyDeviceToHost);

    hipFree(d_bodies);
    hipFree(d_accelerations);
    // time integration
    this->bodies.updatePositionsAndVelocities(this->accelerations, this->dt);
} 
